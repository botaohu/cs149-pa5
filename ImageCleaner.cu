#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif




#if !defined(CUFFT) || CUFFT == 0

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
  // Currently does nothing
}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  CUDA_ERROR_CHECK(hipGetLastError());
  
  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

#else
#include <hipfft/hipfft.h>

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;
  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device

  unsigned int eight = size_y / 8;
  unsigned int eight7 = size_y - eight;
  hipfftHandle plan;
  hipfftComplex *data;
  hipfftPlan2d(&plan, size_x, size_y, HIPFFT_C2C);
  CUDA_ERROR_CHECK(hipMalloc((void**) &data, sizeof(hipfftComplex) * size_x * size_y));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  hipfftComplex *dataLocal = new hipfftComplex[size_x * size_y];
  for (int i = 0; i < size_x * size_y; i++) {
    dataLocal[i].x = real_image[i];
    dataLocal[i].y = imag_image[i];
  }
  CUDA_ERROR_CHECK(hipMemcpy(data, dataLocal, sizeof(hipfftComplex) * size_x * size_y, hipMemcpyHostToDevice));


  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
 
  //----------------------------------------------------------------
  // TODO:  YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  //
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  
  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
  //Filter
  CUDA_ERROR_CHECK(hipMemset2D(data + eight, sizeof(hipfftComplex) * size_y, 0, sizeof(hipfftComplex) * (eight7 - eight), size_x));
  CUDA_ERROR_CHECK(hipMemset2D(data + eight * size_y,  sizeof(hipfftComplex) * size_y, 0, sizeof(hipfftComplex) * size_y, eight7 - eight));
  hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(dataLocal, data, sizeof(hipfftComplex) * size_x * size_y, hipMemcpyDeviceToHost));
  for (int i = 0; i < size_x * size_y; i++) {
    real_image[i] =  dataLocal[i].x / (size_x * size_y);
    imag_image[i] = dataLocal[i].y / (size_x * size_y);
  }

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(data));
  hipfftDestroy(plan);

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}


#endif

